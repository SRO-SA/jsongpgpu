#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iomanip>
#include <x86intrin.h>
#include "hip/hip_runtime_api.h"
#include "DyckNew_Parallel_GPU.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <string.h>
#include "JSON_Parallel_Parser.h"

const char * FILENAMES[]={"./inputs/Long_8.txt", "./inputs/Long_16.txt", "./inputs/Long_32.txt", "./inputs/Long_64.txt"};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

double  step1=0,
        step2=0,
        step3=0,
        step4=0,
        step5=0, 
        step6=0, 
        step7=0, 
        step8=0, 
        scanStep=0, 
        lastStep=0,
        correct1=0,
        correct2=0,
        correct3=0,
        correct4=0,
        program=0;

#define RUNTIMES 50

#define BLOCKSIZE 256
#define FILESCOUNT 4
#define NAMELENGTH 25

#define OPENBRACKET 91
#define CLOSEBRACKET 93
#define OPENBRACE 123
#define CLOSEBRACE 125
#define COMMA 44
#define I 73

#define ROW1 1
#define ROW2 2
#define ROW3 3
#define ROW4 4
#define ROW5 5
#define ROW6 6

bool isCorrect(int strLength, long* input, char* string);

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int print(long* input, int length, int rows){
  for(long i =0; i<rows; i++){
    for(long j=0; j<length; j++){
      std::cout << *(input+j+(i*length)) << ' ';
    }
    std::cout << std::endl;
  }
  return 1;
}

int printString(char* input, int length, int rows){
  for(long i =0; i<rows; i++){
    for(long j=0; j<length; j++){
      std::cout << *(input+j+(i*length)) << ' ';
    }
    std::cout << std::endl;
  }
  return 1;  
}

double runMultipleTimes(double(*func)()){
  double runtime = 0.0;

  for(int i=0; i<RUNTIMES; i++){
    runtime += func();
  }
  runtime = runtime/RUNTIMES;
  step1= step1/RUNTIMES;
  step2= step2/RUNTIMES;
  step3= step3/RUNTIMES;
  step4= step4/RUNTIMES;
  step5= step5/RUNTIMES; 
  step6= step6/RUNTIMES; 
  step7= step7/RUNTIMES; 
  step8= step8/RUNTIMES;
  scanStep= scanStep/RUNTIMES; 
  lastStep= lastStep/RUNTIMES;
  correct1= correct1/RUNTIMES;
  correct2= correct2/RUNTIMES;
  correct3= correct3/RUNTIMES;
  correct4= correct4/RUNTIMES;
  //program= program/RUNTIMES;
  std::cout << "First step mean time for " << RUNTIMES << " number of runs: " << step1 << "ms." << std::endl;
  std::cout << "Second step mean time for " << RUNTIMES << " number of runs: " << step2 << "ms." << std::endl;
  std::cout << "Correctenss First step mean time for " << RUNTIMES << " number of runs: " << correct1 << "ms." << std::endl;
  std::cout << "Correctenss Second step mean time for " << RUNTIMES << " number of runs: " << correct2 << "ms." << std::endl;
  std::cout << "Correctenss Third step mean time for " << RUNTIMES << " number of runs: " << correct3 << "ms." << std::endl;
  std::cout << "Correctenss Fourth step mean time for " << RUNTIMES << " number of runs: " << correct4 << "ms." << std::endl;
  std::cout << "Third step mean time for " << RUNTIMES << " number of runs: " << step3 << "ms." << std::endl;
  std::cout << "Fourth step mean time for " << RUNTIMES << " number of runs: " <<step4 << "ms." << std::endl;
  std::cout << "Fifth step mean time for " << RUNTIMES << " number of runs: " << step5 << "ms." << std::endl;
  std::cout << "Sixth step mean time for " << RUNTIMES << " number of runs: " << step6 << "ms." << std::endl;
  std::cout << "Seventh step mean time for " << RUNTIMES << " number of runs: " << step7 << "ms." << std::endl;
  std::cout << "Eighth step mean time for " << RUNTIMES << " number of runs: " << step8 << "ms." << std::endl;
  std::cout << "Scan step mean time for " << RUNTIMES << " number of runs: " << scanStep << "ms." << std::endl;
  std::cout << "Last step mean time for " << RUNTIMES << " number of runs: " << lastStep << "ms." << std::endl;

  std::cout << "Mean time for " << RUNTIMES << " number of runs: " << runtime << "ms." << std::endl;
  //std::cout << "Internal Mean time for " << RUNTIMES << " number of runs: " << program << "ms." << std::endl;

  step1=0;
  step2=0;
  step3=0;
  step4=0;
  step5=0; 
  step6=0; 
  step7=0; 
  step8=0;
  scanStep=0; 
  lastStep=0;
  correct1=0;
  correct2=0;
  correct3=0;
  correct4=0;
  return runtime;
}

__global__
void inv(long length, long * arr, long * res){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    res[arr[ROW1*length + i]] = i;
  }
}

long *sort(int length, int numBlock, long * arr)
{
  long* cudaArr;
  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr, length*ROW2*sizeof(long), hipMemcpyHostToDevice);
  thrust::device_ptr<long> devArr(cudaArr);
  thrust::stable_sort_by_key(thrust::cuda::par, cudaArr, cudaArr+length, cudaArr+length);
  long *res;
  hipMalloc(&res, length*ROW1*sizeof(long));
  inv<<<numBlock, BLOCKSIZE>>>(length, cudaArr, res);
  hipDeviceSynchronize();
  hipFree(cudaArr);
  return res;
}

__global__
void initialize(int step, int length, char* strArr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride){
    int currentChar = (int) strArr[i];
    if(step == 0){
      if(currentChar == OPENBRACKET || currentChar == OPENBRACE || currentChar == COMMA){
        res[i] = 1;
      }
      else{
        res[i] = 0;
      }
      if(currentChar == OPENBRACKET || currentChar == OPENBRACE){
        res[length + i] = 1;
      }
      else if(currentChar == CLOSEBRACKET || currentChar == CLOSEBRACE){
        res[length + i] = -1;
      }
      else {
        res[length + i] = 0;        
      }
      if(currentChar == OPENBRACE || currentChar == OPENBRACKET || currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET){
        res[ROW2*length + i] = 1;
      }
      else{
        res[ROW2*length + i] = 0;
      }
    }
    if(step == 1){
      if(currentChar == OPENBRACE || currentChar == OPENBRACKET){
        res[i] = 1;
      }
      else{ 
        res[i] = 0;
      }
    }
  }
}

__global__
void changeDepth(int length, char* strArr, char* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int) strArr[i];
    if(currentChar == COMMA){
      while(i>0 && (strArr[i-1] == CLOSEBRACKET || strArr[i-1] == CLOSEBRACE)){
        *(res+i) = *(strArr+i-1);
        *(res+i-1) = (char)currentChar;
        i--;
      }
      *(res+i) = currentChar;
      if(strArr[i-1] == OPENBRACKET || strArr[i-1] == OPENBRACE){
        *(res+i) = (char)I;
      }
    }
    else if(res[i] == 0){
      *(res+i) = *(strArr+i);
    }
  }
}

long findDepthAndCount(int length, int numBlock, long** arr, char * string)
{
  hipMalloc(arr, length*ROW3*sizeof(long));
  initialize<<<numBlock, BLOCKSIZE>>>(0, length, string, *arr);
  hipDeviceSynchronize();
  //gpuErrchk( hipPeekAtLastError() );
  thrust::inclusive_scan(thrust::cuda::par, (*arr), (*arr) + length, (*arr));
  thrust::inclusive_scan(thrust::cuda::par, (*arr) + length, (*arr) + ROW2*length, (*arr) + length);
  thrust::exclusive_scan(thrust::cuda::par, (*arr) + ROW2*length, (*arr) + ROW3*length, (*arr) + ROW2*length);
  long res;
  hipMemcpy(&res, (*arr)+(ROW2*length)-1, sizeof(long), hipMemcpyDeviceToHost);
  if(res == 0){
    return 1;
  }
  return -1;
}

__global__
void extract(int length, int arrLength, char* string, long* arr, char* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int)string[i];
    if(i<length && (currentChar == OPENBRACE || currentChar == OPENBRACKET || currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET)){
      res[arr[i]] = string[i];
    }
  }
}

//INPUT Currectness Check BEGIN
__global__
void countNodesRepititionStep(int length, long* arr, int i, long* res){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long j=index; j<length; j+=stride){
    if(i == -1){
      res[j] = arr[j];
      if(j>0 && arr[j]==arr[j-1]){
        res[length + j] = 1;
      }
      else{
        res[length + j] = 0;
      }
    }
    if(i > -1){
      int pow2 = 1<<i;
      res[j] = arr[j];
      if(j >= pow2){
        if((arr[j] == arr[j - pow2])){
          res[length + j] = arr[length + j - pow2] + arr[length + j];
        }
        else {
          res[length + j] = arr[length + j];
        }
      }
      else{
        res[length + j] = arr[length + j];
      }
    }
  }  
}

long* countNodesRepitition(int length, int numBlock, long* arr)
{
  int nextP2 = length == 1 ? 1 : 1 << (32 - __builtin_clz(length-1));
  long * cudaArr;
  long * cudaRes;
  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMalloc(&cudaRes, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*sizeof(long), hipMemcpyDeviceToDevice);
  int i = -1;

  for(int n = nextP2*2; n>1; n=n>>1){
    countNodesRepititionStep<<<numBlock, BLOCKSIZE>>>(length, cudaArr, i, cudaRes);
    hipDeviceSynchronize();
    hipMemcpy(cudaArr, cudaRes,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);    
    i+=1;
  }
  hipFree(cudaArr);
  return (cudaRes);
}

 __global__
void checkCurrectenss(int length, char* string, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    int currentChar = (int) string[i];
    if(currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET){
      long value = arr[i] - 1;
      long base = i - arr[i];
      while(value>0){
        base--;
        if(arr[base]==0){
          --value;
        }
        else{
          value = value + arr[base] - 1;
          base = base - arr[base];
        }
      }
      int openning = (int) string[base];
      if((currentChar == CLOSEBRACE && openning == OPENBRACE)||(currentChar == CLOSEBRACKET && openning == OPENBRACKET)){
        res[i] = 1;
      }
      else{
        res[i] = 0;
      }
    }
    else{
      res[i] = 1;
    }
  }
}

bool isCorrect(int strLength, long* input, char* string)
{
  clock_t start, end, allStart, allEnd;
  char* h_char_test;
  long* h_long_test;
  allStart = clock();
  int arrLength;
  hipMemcpy(&arrLength, input + strLength - 1, sizeof(long), hipMemcpyDeviceToHost);
  arrLength++;
  int numBlock = ((arrLength) + BLOCKSIZE - 1) / BLOCKSIZE;
  char* res;

  start = clock();
  hipMalloc(&res, arrLength*sizeof(char));
  extract<<<numBlock, BLOCKSIZE>>>(strLength, arrLength, string, input, res);
  hipDeviceSynchronize();
  end = clock();
  correct1 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Curretness First Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_char_test = (char*) malloc(sizeof(char)*arrLength);
  // hipMemcpy(h_char_test, res, sizeof(char)*arrLength, hipMemcpyDeviceToHost);
  // printString(h_char_test, arrLength, ROW1);
  // free(h_char_test);
  // std::cout << "-------------End First Step--------------" << std::endl;

  long* arr;
  start = clock();
  hipMalloc(&arr, arrLength*sizeof(long));
  initialize<<<numBlock, BLOCKSIZE>>>(1, arrLength, res, arr);
  hipDeviceSynchronize();
  //gpuErrchk( hipPeekAtLastError() );

  long* longRes;
  thrust::inclusive_scan(thrust::cuda::par, arr, arr + arrLength, arr);
  end = clock();
  correct2 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Curretness Second Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long*) malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, arr, sizeof(long)*arrLength, hipMemcpyDeviceToHost);
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  // std::cout << "-------------End Second Step--------------" << std::endl;

  start = clock();
  longRes = countNodesRepitition(arrLength, numBlock, arr);
  end = clock();
  correct3 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Curretness Third Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long *)malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, longRes, sizeof(long)*arrLength, hipMemcpyDeviceToHost); 
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  // std::cout << "-------------End Third Step--------------" << std::endl;

  start = clock();
  checkCurrectenss<<<numBlock, BLOCKSIZE>>>(arrLength, res, (longRes+arrLength), arr);
  hipDeviceSynchronize();
  thrust::inclusive_scan(thrust::cuda::par, arr, arr + arrLength, arr);
  end = clock();
  correct4 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Curretness Fourth Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long *)malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, arr, sizeof(long)*arrLength, hipMemcpyDeviceToHost); 
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  // std::cout << "-------------End Fourth Step--------------" << std::endl;

  allEnd = clock();

  hipFree(res);
  hipFree(longRes);
  long isCorrect;
  hipMemcpy(&isCorrect, arr+arrLength-1, sizeof(long), hipMemcpyDeviceToHost);
  hipFree(arr);
  // std::cout << "-------------isCorrect--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(allEnd-allStart)/CLOCKS_PER_SEC)*1000 << std::endl;
  // printf("%d\n", isCorrect == arrLength);
  // std::cout << "-------------End isCorrect--------------" << std::endl;
  return isCorrect == arrLength;
}
//INPUT Currectness Check END

__global__
void reduce(int length, int arrLength, char * string, long * arr, long * res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int)string[i];
    if(i < length && (currentChar == OPENBRACKET || currentChar == OPENBRACE || currentChar == COMMA)){
      if(i != 0) {
        res[arr[i-1]] = arr[length + i - 1];
        res[arrLength + arr[i-1]] = arr[i-1];
      }
      else{
        res[0] = 0;
        res[arrLength] = 0;
      } 
    }
  }
}

long * sortByDepth(int length, int numBlock, long * arr)
{
  long * res;
  long* tmp;
  hipMalloc(&res, length*ROW2*sizeof(long));
  hipMemcpy(res, arr,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
  tmp = sort(length, numBlock, arr);
  hipMemcpy((res+length), tmp, length*ROW1*sizeof(long), hipMemcpyDeviceToDevice);
  hipFree(tmp);
  return res;
}

__global__
void findParents(int length, long * arr, long * res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length-1; i+=stride){
    if(arr[i+1] == arr[i] + 1){
      res[arr[length + i + 1]] = arr[length + i];
    }
  }
}

__global__
void propagateParentsAndCountChildrenStep(int length, long* arr, int i, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long j=index; j<length; j+=stride){
    if( i== -1){
      res[j] = arr[j];
      if(j != 0) res[length + j] = 1;
      else res[length + j] = -1;
    }
    if(i > -1){
      int pow2 = 1<<i;
      if(j >= pow2){
        if(arr[j] == -1 && arr[j - pow2] != -1) {
          res[j] = arr[j - pow2];
        }
        else{
          res[j] = arr[j];
        }
        if(!((arr[j] != -1 && arr[j - pow2] == -1) || (arr[j] != -1 && arr[j - pow2] != -1 && arr[j] != arr[j - pow2]))){
          res[length + j] = arr[length + j - pow2] + arr[length + j];
        }
        else {
          res[length + j] = arr[length + j];
        }
      }
      else{
        res[j] = arr[j];
        res[length + j] = arr[length + j];
      }
    }
  }  
}

long* propagateParentsAndCountChildren(int length, int numBlock, long* arr)
{
  int nextP2 = length == 1 ? 1 : 1 << (32 - __builtin_clz(length-1));
  long * cudaArr;
  long * res;
  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMalloc(&res, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*sizeof(long), hipMemcpyDeviceToDevice);
  int i = -1;
  for(int n = nextP2*2; n>1; n=n>>1){
    propagateParentsAndCountChildrenStep<<<numBlock, BLOCKSIZE>>>(length, cudaArr, i, res);
    hipDeviceSynchronize();
    hipMemcpy(cudaArr, res,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);    
    i+=1;
  }
  hipFree(cudaArr);
  return res;
}

__global__
void childsNumber(int length, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    res[i] = arr[i];
    res[length + i] = arr[length + i];
    res[length*ROW2 + i] = 0;
    if(i == length - 1) res[length*ROW2 + arr[i]] = arr[length + i];
    else if(arr[i] != arr[i + 1] && arr[i] != -1) res[length*ROW2 + arr[i]] = arr[length + i];
    if(i == 0 && length == 1) res[length*ROW2] = 0;
  }
}

__global__
void addOne(int length, long* arr)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  if(index==0) arr[length*ROW3] = 0;
  for(long i=index; i<length; i+=stride){
    arr[length*ROW3 + i] = arr[length*ROW3 + i] + 1;
  }
  if(index==0) arr[length*ROW3] = 0;
}

long * allocate(int length, int numBlock, long* arr)
{
  long * cudaArr;
  hipMalloc(&cudaArr, length*ROW4*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*ROW3*sizeof(long), hipMemcpyDeviceToDevice);
  hipMemcpy(cudaArr+length*ROW3+1, arr+length*ROW2,  (length*ROW1-1)*sizeof(long), hipMemcpyDeviceToDevice);
  addOne<<<numBlock, BLOCKSIZE>>>(length, cudaArr);
  hipDeviceSynchronize();
  thrust::inclusive_scan(thrust::cuda::par, cudaArr+ROW3*length, cudaArr + ROW4*length, cudaArr+ROW3*length);
  return cudaArr;
}

long * scan(int length, long* arr)
{
  long * cudaArr;
  long * res;
  hipMalloc(&cudaArr, length*ROW4*sizeof(long));
  hipMalloc(&res, length*ROW1*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*ROW4*sizeof(long), hipMemcpyHostToDevice); 
  thrust::inclusive_scan(thrust::cuda::par, cudaArr+ROW2*length, cudaArr + ROW3*length, res);
  hipFree(cudaArr);
  return res;  
}

__global__
void generateRes(int length, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;  
  for(long i = index; i<length; i+=stride){
    res[arr[length*ROW3 + i]] = arr[length*ROW2 + i];
    if(arr[i] != -1) res[arr[length*ROW3+ arr[i]]+arr[length+ i]] = arr[length*ROW3+ i];
  }
}


double NewRuntime_Parallel_GPU(char* input, int length) {
  hipProfilerStart();
  int attachedLength = length + 1;
  int numBlock = (attachedLength + BLOCKSIZE - 1) / BLOCKSIZE;
  long* res;
  long* fakeRes;
  long* arr;
  char* attacheArr;
  clock_t start, end, allStart, allEnd;
  char* h_char_test;
  long* h_long_test;
  allStart = clock();

  start = clock();

  attacheArr = (char*) malloc(sizeof(char)*attachedLength);
  memcpy(attacheArr, input, length*sizeof(char));
  attacheArr[length] = ',';
  char* d_attacheArr;
  hipMalloc(&d_attacheArr, attachedLength*sizeof(char));
  hipMemcpy(d_attacheArr, attacheArr, attachedLength*sizeof(char), hipMemcpyHostToDevice);
  
  //hipMallocManaged(&attacheArr, attachedLength*sizeof(char));
  //hipMemcpy(attacheArr, input, length*sizeof(char), hipMemcpyHostToDevice);
  //attacheArr[length] = ',';
  char* d_sameDepthArr;
  hipMalloc(&d_sameDepthArr, attachedLength*sizeof(char));
  hipMemcpy(d_sameDepthArr, attacheArr, attachedLength*sizeof(char), hipMemcpyHostToDevice);

  changeDepth<<<numBlock, BLOCKSIZE>>>(attachedLength, d_attacheArr, d_sameDepthArr);
  hipDeviceSynchronize();
  free(attacheArr);
  hipFree(d_attacheArr);
  end = clock();
  step1 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------First Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_char_test = (char*)malloc(sizeof(char)*attachedLength);
  // hipMemcpy(h_char_test, d_sameDepthArr, sizeof(char)*attachedLength, hipMemcpyDeviceToHost);
  // printString(h_char_test, attachedLength, ROW1);
  // free(h_char_test);
  // std::cout << "-------------End First Step--------------" << std::endl;

  start = clock();
  long *d_arr;
  long correctDepth;
  correctDepth = findDepthAndCount(attachedLength, numBlock, &d_arr, d_sameDepthArr);
  end = clock();
  step2 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Second Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long*)malloc(sizeof(long)*attachedLength*ROW3);
  // hipMemcpy(h_long_test, d_arr, sizeof(long)*attachedLength*ROW3, hipMemcpyDeviceToHost);
  // print(h_long_test, attachedLength, ROW3);
  // free(h_long_test);
  // std::cout << "-------------End Second Step--------------" << std::endl;
  long arrLength;
  hipMemcpy(&arrLength, d_arr+(attachedLength-1), sizeof(long), hipMemcpyDeviceToHost);
  if(correctDepth != -1){
    bool correct;
    correct = isCorrect(attachedLength, d_arr+(attachedLength)*ROW2, d_sameDepthArr);
    if(correct){      
      start = clock();
      hipMalloc(&arr, attachedLength*ROW4*sizeof(long));
      hipMalloc(&res, arrLength*ROW4*sizeof(long));
      hipMemcpy(arr, d_arr,  attachedLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      reduce<<<numBlock, BLOCKSIZE>>>(attachedLength, arrLength, d_sameDepthArr, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step3 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Third Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW2);
      // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW2);
      // free(h_long_test);
      // std::cout << "-------------End Third Step--------------" << std::endl;
      hipFree(d_sameDepthArr);
      hipFree(d_arr);
      int numBlock = (arrLength + BLOCKSIZE - 1) / BLOCKSIZE;

      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW2*sizeof(long), hipMemcpyHostToHost);
      fakeRes = sortByDepth(arrLength, numBlock, arr);
      end = clock();
      step4 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Fourth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long *)malloc(sizeof(long)*arrLength*ROW2);
      // hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW2);
      // free(h_long_test);
      // std::cout << "-------------End Fourth Step--------------" << std::endl;

      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      hipFree(fakeRes);
      hipMemset(res, -1, arrLength*ROW1*sizeof(long));
      findParents<<<numBlock, BLOCKSIZE>>>( arrLength, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step5 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Fifth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW1);
      // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW1, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW1);
      // free(h_long_test);
      // std::cout << "-------------End Fifth Step--------------" << std::endl;
      
      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW1*sizeof(long), hipMemcpyDeviceToDevice);
      fakeRes = propagateParentsAndCountChildren(arrLength, numBlock, arr);
      end = clock();
      step6 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Sixth Step--------------" << std::endl;      
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*)malloc(sizeof(long)*arrLength*ROW2);
      // hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW2);
      // free(h_long_test);
      // std::cout << "-------------End Sixth Step--------------" << std::endl;

      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      hipFree(fakeRes);
      hipMemset(res, -1, arrLength*ROW3*sizeof(long));
      childsNumber<<<numBlock, BLOCKSIZE>>>(arrLength, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step7 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Seventh Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW3);
      // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW3, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW3);
      // free(h_long_test);
      // std::cout << "-------------End Seventh Step--------------" << std::endl;
      
      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW3*sizeof(long), hipMemcpyDeviceToDevice);
      fakeRes = allocate(arrLength, numBlock, arr);
      end = clock();
      step8 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Eighth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW4);
      // hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW4, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW4);
      // free(h_long_test);
      // std::cout << "-------------End Eighth Step--------------" << std::endl;

      long* sumRes;
      hipMalloc(&sumRes, arrLength*ROW1*sizeof(long));
      start = clock();
      sumRes = scan(arrLength, fakeRes);
      end = clock();
      scanStep += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Scan Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW1);
      // hipMemcpy(h_long_test, sumRes, sizeof(long)*arrLength*ROW1, hipMemcpyDeviceToHost);
      // print(h_long_test, arrLength, ROW1);
      // free(h_long_test);
      // std::cout << "-------------End Scan Step--------------" << std::endl;
      long resLength;
      hipMemcpy(&resLength, sumRes + arrLength - 1, sizeof(long), hipMemcpyDeviceToHost);
      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW4*sizeof(long), hipMemcpyHostToHost);
      hipFree(sumRes);
      hipFree(fakeRes);
      hipFree(res);
      hipMalloc(&res, (arrLength+resLength)*sizeof(long));
      hipMemset(res, 0, (arrLength+resLength)*sizeof(long));
      generateRes<<<numBlock, BLOCKSIZE>>>(arrLength,  arr, res);
      hipDeviceSynchronize();
      end = clock();
      lastStep += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Last Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // h_long_test = (long*) malloc(sizeof(long)*(arrLength+resLength)*ROW1);
      // hipMemcpy(h_long_test, res, sizeof(long)*(arrLength+resLength)*ROW1, hipMemcpyDeviceToHost);
      // print(h_long_test, (arrLength+resLength), ROW1);
      // free(h_long_test);
      // std::cout << "-------------End Last Step--------------" << std::endl;
      hipFree(arr);
      hipFree(res);
      allEnd = clock();
      hipProfilerStop();    
      //*******************************//
      // size_t l_free = 0;
      // size_t l_Total = 0;
      // hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
      // size_t allocated = (l_Total - l_free);
      // std::cout << "Total: " << l_Total << " Free: " << l_free << " Allocated: " << allocated << std::endl;
      //*******************************//
      //program += ((double)(allEnd-allStart)/CLOCKS_PER_SEC)*1000;
      //printf("program: %f\n", program);

    } 
    else{
      printf("Input wrong\n");
      return 0;  
    }
  }
  else {
    printf("Input invalid\n");
    return 0;
  }
  return (double)(allEnd-allStart);
}

char **loadMultipleFiles(int length, char** names, int * filesLength){
  char** texts = (char **)malloc(length*sizeof(char *));
  for(int i = 0; i< length; i++){
    FILE * f = fopen(names[i], "r");
    if(f){
      fseek(f, 0, SEEK_END);
      filesLength[i] = ftell(f);
      fseek(f, 0, SEEK_SET);
      texts[i] = (char *)malloc(sizeof(char) * filesLength[i]);
      if(texts[i]) {
          fread(texts[i], 1, filesLength[i], f);
      }
      fclose(f);
    }
  }
  return texts;
}

char* loadFile(int* fileLength){
  char * input = 0;
  long length;
  // Long input
  //FILE * f = fopen("./inputs/Long.txt", "r");

  // Long 2 times
  //FILE * f = fopen("./inputs/Long_2.txt", "r");

  // Long 4 times
  //FILE * f = fopen("./inputs/Long_4.txt", "r");

  // Long 8 times
  //FILE * f = fopen("./inputs/Long_8.txt", "r");

  // Long 16 times
  //FILE * f = fopen("./inputs/Long_16.txt", "r");

  // Long 32 times
  //FILE * f = fopen("./inputs/Long_32.txt", "r");

  // Long 64 times
  //FILE * f = fopen("./inputs/Long_64.txt", "r");

  // Long 128 times
  FILE * f = fopen("./inputs/Long_128.txt", "r");

  // Long 256 times
  //FILE * f = fopen("./inputs/Long_256.txt", "r");

  // Long 512 times
  //FILE * f = fopen("./inputs/Long_512.txt", "r");

  // Long 1024 times
  //FILE * f = fopen("./inputs/Long_1024.txt", "r");

  // Base input
  //FILE * f = fopen("./inputs/Base.txt", "r");

  // Author input
  //FILE * f = fopen("./inputs/Author.txt", "r");
  
  // False input
  //FILE * f = fopen("./inputs/False.txt", "r");
  
  // One input
  //FILE * f = fopen("./inputs/One.txt", "r");

  if(f){
    fseek(f, 0, SEEK_END);
    length = ftell(f);
    fseek(f, 0, SEEK_SET);
    input = (char *)malloc(sizeof(char) * length);
    if(input) {
        fread(input, 1, length, f);
    }
    fclose(f);
    *fileLength = length;
    return input;
  }
  return 0;
}

double batchMode()
{
  char ** texts;
  char ** names = (char**) malloc(FILESCOUNT*sizeof(char *));
  int * fileLength = (int *)malloc(FILESCOUNT*sizeof(int));
  for(int i = 0; i< FILESCOUNT; i++){
    names[i] = (char *)malloc(NAMELENGTH*sizeof(char));
    strcpy(names[i], FILENAMES[i]);
  }
  texts = loadMultipleFiles(FILESCOUNT, names, fileLength);
  double GPUparallelTime = 0;
  //NewRuntime_Parallel_GPU(texts[0], fileLength[0]);
  if(texts){
    for(int i=0; i<FILESCOUNT; i++){
      //std::cout <<  fileLength[i] << std::endl;
      GPUparallelTime += NewRuntime_Parallel_GPU(texts[i], fileLength[i]);
      //std::cout << "Parallel GPU time elapsed: " << std::setprecision (17) << (GPUparallelTime/CLOCKS_PER_SEC)*1000 << "ms." << std::endl;
    }
    return (GPUparallelTime/CLOCKS_PER_SEC)*1000;
  }
  else{
    printf("Cannot read file\n");
    return -1;
  }
}

double singleMode(){
  int length;
  char* input = loadFile(&length);
  if(input){
    //std::cout <<  length << std::endl;
    double GPUparallelTime = 0;
    //NewRuntime_Parallel_GPU(input, length);
    GPUparallelTime = NewRuntime_Parallel_GPU(input, length);
    //std::cout << "Parallel GPU time elapsed: " << std::setprecision (17) << (GPUparallelTime/CLOCKS_PER_SEC)*1000 << "ms." << std::endl;
    return (GPUparallelTime/CLOCKS_PER_SEC)*1000;
  }
  else{
    printf("Cannot read file\n");
    return -1;
  }
}

int main(int argc, char **argv)
{
  double result;
  if (argv[1] != NULL && strcmp(argv[1], "-b") == 0){
    std::cout << "Batch mode..." << std::endl;
    result = runMultipleTimes(batchMode);
  }
  else{
    std::cout << "Single mode..." << std::endl;
    result = runMultipleTimes(singleMode);
  }
  return 0;
}