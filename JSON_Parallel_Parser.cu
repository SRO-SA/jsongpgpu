#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iomanip>
#include <x86intrin.h>
#include "hip/hip_runtime_api.h"
#include "DyckNew_Parallel_GPU.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <string.h>
#include "JSON_Parallel_Parser.h"

struct not_zero
{
    __host__ __device__
    bool operator()(const int x)
    {
        return (x > 0);
    }
};

struct is_minus
{
    __host__ __device__
    bool operator()(const int x)
    {
        return (x < 0);
    }
};


const char * FILENAMES[]={"./inputs/Long_8.txt", "./inputs/Long_16.txt", "./inputs/Long_32.txt", "./inputs/Long_64.txt"};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

double  step1=0,
        step2=0,
        step3=0,
        step4=0,
        step5=0, 
        step6=0, 
        step7=0, 
        step8=0, 
        scanStep=0, 
        lastStep=0,
        correct1=0,
        correct2=0,
        correct3=0,
        correct4=0,
        program=0;

#define RUNTIMES 50

#define BLOCKSIZE 256
#define FILESCOUNT 4
#define NAMELENGTH 25

#define OPENBRACKET 91
#define CLOSEBRACKET 93
#define OPENBRACE 123
#define CLOSEBRACE 125
#define COMMA 44
#define I 73

#define ROW1 1
#define ROW2 2
#define ROW3 3
#define ROW4 4
#define ROW5 5
#define ROW6 6

bool isCorrect(int strLength, long* input, char* string);

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int print(long* input, int length, int rows){
  for(long i =0; i<rows; i++){
    for(long j=0; j<length; j++){
      std::cout << *(input+j+(i*length)) << ' ';
    }
    std::cout << std::endl;
  }
  return 1;
}

int printString(char* input, int length, int rows){
  for(long i =0; i<rows; i++){
    for(long j=0; j<length; j++){
      std::cout << *(input+j+(i*length)) << ' ';
    }
    std::cout << std::endl;
  }
  return 1;  
}

double runMultipleTimes(double(*func)()){
  double runtime = 0.0;

  for(int i=0; i<RUNTIMES; i++){
    runtime += func();
  }
  runtime = runtime/RUNTIMES;
  step1= step1/RUNTIMES;
  step2= step2/RUNTIMES;
  step3= step3/RUNTIMES;
  step4= step4/RUNTIMES;
  step5= step5/RUNTIMES; 
  step6= step6/RUNTIMES; 
  step7= step7/RUNTIMES; 
  step8= step8/RUNTIMES;
  scanStep= scanStep/RUNTIMES; 
  lastStep= lastStep/RUNTIMES;
  correct1= correct1/RUNTIMES;
  correct2= correct2/RUNTIMES;
  correct3= correct3/RUNTIMES;
  correct4= correct4/RUNTIMES;
  //program= program/RUNTIMES;
  std::cout << "First step mean time for " << RUNTIMES << " number of runs: " << step1 << "ms." << std::endl;
  std::cout << "Second step mean time for " << RUNTIMES << " number of runs: " << step2 << "ms." << std::endl;
  std::cout << "Correctenss First step mean time for " << RUNTIMES << " number of runs: " << correct1 << "ms." << std::endl;
  std::cout << "Correctenss Second step mean time for " << RUNTIMES << " number of runs: " << correct2 << "ms." << std::endl;
  std::cout << "Correctenss Third step mean time for " << RUNTIMES << " number of runs: " << correct3 << "ms." << std::endl;
  std::cout << "Correctenss Fourth step mean time for " << RUNTIMES << " number of runs: " << correct4 << "ms." << std::endl;
  std::cout << "Third step mean time for " << RUNTIMES << " number of runs: " << step3 << "ms." << std::endl;
  std::cout << "Fourth step mean time for " << RUNTIMES << " number of runs: " <<step4 << "ms." << std::endl;
  std::cout << "Fifth step mean time for " << RUNTIMES << " number of runs: " << step5 << "ms." << std::endl;
  std::cout << "Sixth step mean time for " << RUNTIMES << " number of runs: " << step6 << "ms." << std::endl;
  std::cout << "Seventh step mean time for " << RUNTIMES << " number of runs: " << step7 << "ms." << std::endl;
  std::cout << "Eighth step mean time for " << RUNTIMES << " number of runs: " << step8 << "ms." << std::endl;
  std::cout << "Scan step mean time for " << RUNTIMES << " number of runs: " << scanStep << "ms." << std::endl;
  std::cout << "Last step mean time for " << RUNTIMES << " number of runs: " << lastStep << "ms." << std::endl;

  std::cout << "Mean time for " << RUNTIMES << " number of runs: " << runtime << "ms." << std::endl;
  //std::cout << "Internal Mean time for " << RUNTIMES << " number of runs: " << program << "ms." << std::endl;

  step1=0;
  step2=0;
  step3=0;
  step4=0;
  step5=0; 
  step6=0; 
  step7=0; 
  step8=0;
  scanStep=0; 
  lastStep=0;
  correct1=0;
  correct2=0;
  correct3=0;
  correct4=0;
  return runtime;
}

__global__
void inv(long length, long * arr, long * res){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    res[arr[ROW1*length + i]] = i;
  }
}

long *sort(int length, int numBlock, long * arr)
{
  clock_t start, end;
  double total = 0;
  long* cudaArr;
  start = clock();
  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr, length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
  thrust::device_ptr<long> devArr(cudaArr);
  end = clock();

  //total = ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  //printf("allocation time: %f\n", total);

  start = clock();
  thrust::stable_sort_by_key(thrust::cuda::par, cudaArr, cudaArr+length, cudaArr+length);
  end = clock();

  //total = ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  //printf("sort time: %f\n", total);

  long *res;
  hipMalloc(&res, length*ROW1*sizeof(long));
  start = clock();
  inv<<<numBlock, BLOCKSIZE>>>(length, cudaArr, res);
  hipDeviceSynchronize();
  end = clock();

  //total = ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  //printf("inverse time: %f\n", total);
  
  hipFree(cudaArr);
  return res;
}

__global__
void initialize(int step, int length, char* strArr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride){
    int currentChar = (int) strArr[i];
    if(step == 0){
      if(currentChar == OPENBRACKET || currentChar == OPENBRACE || currentChar == COMMA){
        res[i] = 1;
      }
      else{
        res[i] = 0;
      }
      if(currentChar == OPENBRACKET || currentChar == OPENBRACE){
        res[length + i] = 1;
      }
      else if(currentChar == CLOSEBRACKET || currentChar == CLOSEBRACE){
        res[length + i] = -1;
      }
      else {
        res[length + i] = 0;        
      }
      if(currentChar == OPENBRACE || currentChar == OPENBRACKET || currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET){
        res[ROW2*length + i] = 1;
      }
      else{
        res[ROW2*length + i] = 0;
      }
    }
    if(step == 1){
      if(currentChar == OPENBRACE || currentChar == OPENBRACKET){
        res[i] = 1;
      }
      else{ 
        res[i] = 0;
      }
    }
  }
}

__global__
void changeDepth(int length, char* strArr, char* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int) strArr[i];
    if(currentChar == COMMA){
      while(i>0 && (strArr[i-1] == CLOSEBRACKET || strArr[i-1] == CLOSEBRACE)){
        *(res+i) = *(strArr+i-1);
        *(res+i-1) = (char)currentChar;
        i--;
      }
      *(res+i) = currentChar;
      if(strArr[i-1] == OPENBRACKET || strArr[i-1] == OPENBRACE){
        *(res+i) = (char)I;
      }
    }
    else if(res[i] == 0){
      *(res+i) = *(strArr+i);
    }
  }
}

long findDepthAndCount(int length, int numBlock, long** arr, char * string)
{
  hipMalloc(arr, length*ROW3*sizeof(long));
  initialize<<<numBlock, BLOCKSIZE>>>(0, length, string, *arr);
  hipDeviceSynchronize();
  //gpuErrchk( hipPeekAtLastError() );
  thrust::inclusive_scan(thrust::cuda::par, (*arr), (*arr) + length, (*arr));
  thrust::inclusive_scan(thrust::cuda::par, (*arr) + length, (*arr) + ROW2*length, (*arr) + length);
  thrust::exclusive_scan(thrust::cuda::par, (*arr) + ROW2*length, (*arr) + ROW3*length, (*arr) + ROW2*length);
  long res;
  hipMemcpy(&res, (*arr)+(ROW2*length)-1, sizeof(long), hipMemcpyDeviceToHost);
  if(res == 0){
    return 1;
  }
  return -1;
}

__global__
void extract(int length, int arrLength, char* string, long* arr, char* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int)string[i];
    if(i<length && (currentChar == OPENBRACE || currentChar == OPENBRACKET || currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET)){
      res[arr[i]] = string[i];
    }
  }
}

//INPUT Currectness Check BEGIN
__global__
void countNodesRepititionStep(int length, long* arr, int i, long* res){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long j=index; j<length; j+=stride){
    if(i == -1){
      res[j] = arr[j];
      if(j>0 && arr[j]==arr[j-1]){
        res[length + j] = 1;
      }
      else{
        res[length + j] = 0;
      }
    }
    if(i > -1){
      int pow2 = 1<<i;
      res[j] = arr[j];
      if(j >= pow2){
        if((arr[j] == arr[j - pow2])){
          res[length + j] = arr[length + j - pow2] + arr[length + j];
        }
        else {
          res[length + j] = arr[length + j];
        }
      }
      else{
        res[length + j] = arr[length + j];
      }
    }
  }  
}

long* countNodesRepitition(int length, int numBlock, long* arr)
{
  int nextP2 = length == 1 ? 1 : 1 << (32 - __builtin_clz(length-1));
  long * cudaArr;
  long * cudaRes;
  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMalloc(&cudaRes, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*sizeof(long), hipMemcpyDeviceToDevice);
  int i = -1;

  for(int n = nextP2*2; n>1; n=n>>1){
    countNodesRepititionStep<<<numBlock, BLOCKSIZE>>>(length, cudaArr, i, cudaRes);
    hipDeviceSynchronize();
    hipMemcpy(cudaArr, cudaRes,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);    
    i+=1;
  }
  hipFree(cudaArr);
  return (cudaRes);
}

 __global__
void checkCurrectenss(int length, char* string, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    int currentChar = (int) string[i];
    if(currentChar == CLOSEBRACE || currentChar == CLOSEBRACKET){
      long value = arr[i] - 1;
      long base = i - arr[i];
      while(value>0){
        base--;
        if(arr[base]==0){
          --value;
        }
        else{
          value = value + arr[base] - 1;
          base = base - arr[base];
        }
      }
      int openning = (int) string[base];
      if((currentChar == CLOSEBRACE && openning == OPENBRACE)||(currentChar == CLOSEBRACKET && openning == OPENBRACKET)){
        res[i] = 1;
      }
      else{
        res[i] = 0;
      }
    }
    else{
      res[i] = 1;
    }
  }
}

__global__
void set_open_odd_close_even(char* input_d, uint32_t* o_o_c_e, uint32_t* o_e_c_o, int length){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    char currentChar = input_d[i];
    o_o_c_e[i] = ((i & 1) == ((currentChar & 2) >> 1)); // odd opening and even closing (== works as XNOR)
    o_e_c_o[i] = ((i & 1) ^  ((currentChar & 2) >> 1)); // even opening and odd closing
  }
}

__global__
void check_is_matched(char* input_d, uint8_t* res_check, int length){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    char currentChar = input_d[i*2];
    char nextChar = input_d[i*2+1];
      uint8_t sixth_bit_i = (currentChar >> 5) & 1;
      uint8_t second_bit_i = (currentChar >> 1) & 1;
      uint8_t sixth_bit_i_1 = (nextChar >> 5) & 1;
      uint8_t second_bit_i_1 = (nextChar >> 1) & 1; 

      res_check[i] = (sixth_bit_i == sixth_bit_i_1) && (second_bit_i == 1) && (second_bit_i ^ second_bit_i_1) ? 0 : 1;
      //matched &= (sixth_bit_i == sixth_bit_i_1) && (second_bit_i == 1) && (second_bit_i ^ second_bit_i_1);    

  }
}


bool matching(char *input_d, int length, int iter, int numBlock){
  bool matched = true;
  if(length < 3 || iter == 1){
    uint8_t* res_check;
    int length_divided = length/2;
    int dividedNumBlock = ((length_divided) + BLOCKSIZE - 1) / BLOCKSIZE;

    hipMalloc(&res_check, sizeof(uint8_t)*length_divided);
    check_is_matched<<<dividedNumBlock, BLOCKSIZE>>>(input_d, res_check, length_divided);
    matched = thrust::reduce(thrust::cuda::par, res_check, res_check+length_divided) == 0 ? true : false;
    hipFree(res_check);
    // char * res_check = (char *)malloc(sizeof(char)*length);
    // hipMemcpy(res_check, input_d, sizeof(char)*length, hipMemcpyDeviceToHost);
    // for(int i = 0; i< length; i+=2){
    //   uint8_t sixth_bit_i = (res_check[i] >> 5) & 1;
    //   uint8_t second_bit_i = (res_check[i] >> 1) & 1;
    //   uint8_t sixth_bit_i_1 = (res_check[i+1] >> 5) & 1;
    //   uint8_t second_bit_i_1 = (res_check[i+1] >> 1) & 1; 

    //   matched &= (sixth_bit_i == sixth_bit_i_1) && (second_bit_i == 1) && (second_bit_i ^ second_bit_i_1);    

    // }
    // free(res_check);

    // if(!matched){
    //   std::cout << "-------------Curretness "<< iter << " Step--------------" << std::endl;
    //   //std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
    //   char * h_char_test = (char*) malloc(sizeof(char)*length);
    //   hipMemcpy(h_char_test, input_d, sizeof(char)*length, hipMemcpyDeviceToHost);
    //   printString(h_char_test, length, ROW1);
    //   free(h_char_test);
    //   std::cout << "-------------End "<< iter <<" Step--------------" << std::endl;
    
    // }
    return matched;
  }
  uint32_t * o_o_c_e;

  uint32_t * o_e_c_o;


  hipMalloc(&o_o_c_e, sizeof(uint32_t)*length);
  hipMalloc(&o_e_c_o, sizeof(uint32_t)*length);


  set_open_odd_close_even<<<numBlock, BLOCKSIZE>>>(input_d, o_o_c_e, o_e_c_o, length);
  hipDeviceSynchronize();

  char * right_reduced;
  char * left_reduced;
  uint32_t right_length;
  uint32_t left_length;

  left_length = thrust::count_if(thrust::cuda::par, o_o_c_e, o_o_c_e+length, not_zero());
  right_length = thrust::count_if(thrust::cuda::par, o_e_c_o, o_e_c_o+length, not_zero());
  // printf("left: %d\n", left_length);
  // printf("right: %d\n", right_length);

  if(left_length == 0 || right_length == 0){
    uint8_t* res_check;
    int length_divided = length/2;
    int dividedNumBlock = ((length_divided) + BLOCKSIZE - 1) / BLOCKSIZE;
    hipMalloc(&res_check, sizeof(uint8_t)*length_divided);
    check_is_matched<<<dividedNumBlock, BLOCKSIZE>>>(input_d, res_check, length_divided);
    matched = thrust::reduce(thrust::cuda::par, res_check, res_check+length_divided) == 0 ? true : false;
    hipFree(res_check);
    hipFree(o_e_c_o);
    hipFree(o_o_c_e);

    // char * res_check = (char *)malloc(sizeof(char)*length);
    // hipMemcpy(res_check, input_d, sizeof(char)*length, hipMemcpyDeviceToHost);
    // for(int i = 0; i< length; i+=2){
    //   uint8_t sixth_bit_i = (res_check[i] >> 5) & 1;
    //   uint8_t second_bit_i = (res_check[i] >> 1) & 1;
    //   uint8_t sixth_bit_i_1 = (res_check[i+1] >> 5) & 1;
    //   uint8_t second_bit_i_1 = (res_check[i+1] >> 1) & 1; 

    //   matched &= (sixth_bit_i == sixth_bit_i_1) && (second_bit_i == 1) && (second_bit_i ^ second_bit_i_1);

    // }
    // free(res_check);
    // if(!matched){
    //   std::cout << "-------------Curretness "<< iter << " Step--------------" << std::endl;
    //   //std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
    //   char * h_char_test = (char*) malloc(sizeof(char)*length);
    //   hipMemcpy(h_char_test, input_d, sizeof(char)*length, hipMemcpyDeviceToHost);
    //   printString(h_char_test, length, ROW1);
    //   free(h_char_test);
    //   std::cout << "-------------End "<< iter <<" Step--------------" << std::endl;
    
    // }
    return matched;
  }

  //hipMemcpy(&right_length, o_e_c_o+length-1, sizeof(uint32_t), hipMemcpyDeviceToHost);
  //hipMemcpy(&left_length, o_o_c_e+length-1, sizeof(uint32_t), hipMemcpyDeviceToHost);



  hipMalloc(&right_reduced, sizeof(char)*right_length);
  hipMalloc(&left_reduced, sizeof(char)*left_length);

  thrust::copy_if(thrust::cuda::par, input_d, input_d+length, o_e_c_o, right_reduced, not_zero());
  thrust::copy_if(thrust::cuda::par, input_d, input_d+length, o_o_c_e, left_reduced, not_zero());
  bool right_res;
  bool left_res;

  hipFree(o_e_c_o);
  hipFree(o_o_c_e);

  right_res = matching(right_reduced, right_length, iter>>1, ((length) + BLOCKSIZE - 1) / BLOCKSIZE);
  left_res = matching(left_reduced, left_length, iter >> 1,  ((length) + BLOCKSIZE - 1) / BLOCKSIZE);

  hipFree(right_reduced);
  hipFree(left_reduced);

  return (right_res && left_res);
}

bool isCorrect(int strLength, long* input, char* string)
{
  clock_t start, end, allStart, allEnd;
  char* h_char_test;
  long* h_long_test;
  allStart = clock();
  int arrLength;
  hipMemcpy(&arrLength, input + strLength - 1, sizeof(long), hipMemcpyDeviceToHost);
  arrLength++;
  int numBlock = ((arrLength) + BLOCKSIZE - 1) / BLOCKSIZE;
  char* res;

  start = clock();
  hipMalloc(&res, arrLength*sizeof(char));
  extract<<<numBlock, BLOCKSIZE>>>(strLength, arrLength, string, input, res);
  hipDeviceSynchronize();
  end = clock();
  correct1 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Curretness First Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // printf("%d\n", arrLength);
  // //h_char_test = (char*) malloc(sizeof(char)*arrLength);
  // //hipMemcpy(h_char_test, res, sizeof(char)*arrLength, hipMemcpyDeviceToHost);
  // //printString(h_char_test, arrLength, ROW1);
  // //free(h_char_test);
  // std::cout << "-------------End First Step--------------" << std::endl;

  int nextP2 = arrLength == 1 ? 1 : 1 << (32 - __builtin_clz(arrLength-1));

  start = clock();
  bool isCorrect =  matching(res, arrLength, nextP2, numBlock);
  end = clock();

  allEnd = clock();
  hipFree(res);
  // std::cout << "-------------isCorrect--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(allEnd-allStart)/CLOCKS_PER_SEC)*1000 << std::endl;
  // printf("%d\n", isCorrect);
  // std::cout << "-------------End isCorrect--------------" << std::endl;

  return isCorrect;
  
  /*
  long* arr;
  start = clock();
  hipMalloc(&arr, arrLength*sizeof(long));
  initialize<<<numBlock, BLOCKSIZE>>>(1, arrLength, res, arr);
  hipDeviceSynchronize();
  //gpuErrchk( hipPeekAtLastError() );

  long* longRes;
  thrust::inclusive_scan(thrust::cuda::par, arr, arr + arrLength, arr);
  end = clock();
  correct2 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  std::cout << "-------------Curretness Second Step--------------" << std::endl;
  std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long*) malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, arr, sizeof(long)*arrLength, hipMemcpyDeviceToHost);
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  std::cout << "-------------End Second Step--------------" << std::endl;

  start = clock();
  longRes = countNodesRepitition(arrLength, numBlock, arr);
  end = clock();
  correct3 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  std::cout << "-------------Curretness Third Step--------------" << std::endl;
  std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long *)malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, longRes, sizeof(long)*arrLength, hipMemcpyDeviceToHost); 
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  std::cout << "-------------End Third Step--------------" << std::endl;

  start = clock();
  checkCurrectenss<<<numBlock, BLOCKSIZE>>>(arrLength, res, (longRes+arrLength), arr);
  hipDeviceSynchronize();
  thrust::inclusive_scan(thrust::cuda::par, arr, arr + arrLength, arr);
  end = clock();
  correct4 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  std::cout << "-------------Curretness Fourth Step--------------" << std::endl;
  std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // h_long_test = (long *)malloc(sizeof(long)*arrLength);
  // hipMemcpy(h_long_test, arr, sizeof(long)*arrLength, hipMemcpyDeviceToHost); 
  // print(h_long_test, arrLength, ROW1);
  // free(h_long_test);
  std::cout << "-------------End Fourth Step--------------" << std::endl;

  allEnd = clock();

  hipFree(res);
  hipFree(longRes);
  long isCorrect;
  hipMemcpy(&isCorrect, arr+arrLength-1, sizeof(long), hipMemcpyDeviceToHost);
  hipFree(arr);
  std::cout << "-------------isCorrect--------------" << std::endl;
  std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(allEnd-allStart)/CLOCKS_PER_SEC)*1000 << std::endl;
  // printf("%d\n", isCorrect == arrLength);
  std::cout << "-------------End isCorrect--------------" << std::endl;
  return isCorrect == arrLength;
  */
}
//INPUT Currectness Check END

__global__
void reduce(int length, int arrLength, char * string, long * arr, long * res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i = index; i< length; i+=stride)
  {
    int currentChar = (int)string[i];
    if(i < length && (currentChar == OPENBRACKET || currentChar == OPENBRACE || currentChar == COMMA)){
      if(i != 0) {
        res[arr[i-1]] = arr[length + i - 1];
        res[arrLength + arr[i-1]] = arr[i-1];
      }
      else{
        res[0] = 0;
        res[arrLength] = 0;
      } 
    }
  }
}

long * sortByDepth(int length, int numBlock, long * arr)
{
  long * res;
  long* tmp;
  hipMalloc(&res, length*ROW2*sizeof(long));
  hipMemcpy(res, arr,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
  tmp = sort(length, numBlock, arr);
  hipMemcpy((res+length), tmp, length*ROW1*sizeof(long), hipMemcpyDeviceToDevice);
  hipFree(tmp);
  return res;
}

__global__
void findParents(int length, long * arr, long * res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length-1; i+=stride){
    if(arr[i+1] == arr[i] + 1){
      res[arr[length + i + 1]] = arr[length + i];
    }
  }
}

__global__
void propagateParentsAndCountChildrenStep(int length, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long j=index; j<length; j+=stride){
    res[j] = arr[j];
    if(j != 0) res[length + j] = 1;
    else res[length + j] = -1;
  }  
}

__global__
void propagateParents(int length, long* arr, long* keys, long* res){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    res[i] = (keys[i] < 0 && i>0) ? arr[arr[i]] : arr[i];
  } 

}

long* propagateParentsAndCountChildren(int length, int numBlock, long* arr)
{
  int nextP2 = length == 1 ? 1 : 1 << (32 - __builtin_clz(length-1));
  //clock_t start, end;
  long * cudaArr;
  long * keys_d;
  long * res;
  long * index_d;
  thrust::plus<int> op;
  long first_index = -1;

  // long * h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, arr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);

  hipMalloc(&cudaArr, length*ROW2*sizeof(long));
  hipMalloc(&keys_d, length*ROW1*sizeof(long));
  hipMalloc(&index_d, length*ROW1*sizeof(long));

  hipMalloc(&res, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*sizeof(long), hipMemcpyDeviceToDevice);
  hipMemcpy(keys_d, arr,  length*sizeof(long), hipMemcpyDeviceToDevice);

  int i = -1;
  propagateParentsAndCountChildrenStep<<<numBlock, BLOCKSIZE>>>(length, cudaArr, cudaArr);  
  hipDeviceSynchronize();

  // long* h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, cudaArr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);

  thrust::inclusive_scan_by_key(thrust::cuda::par, keys_d, keys_d+length, cudaArr, cudaArr);

  // long* h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, cudaArr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);
  
  hipMemcpy(keys_d, cudaArr,  length*sizeof(long), hipMemcpyDeviceToDevice);
  thrust::sequence(thrust::cuda::par, index_d, index_d+length);

  thrust::transform_if(thrust::cuda::par, cudaArr, cudaArr+length, index_d, keys_d, cudaArr, op, is_minus());
  // long* h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, cudaArr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);

  // *(res+i) = *(cudaArr+*(cudaArr+i)) if key+i < 0
  propagateParents<<<numBlock, BLOCKSIZE>>>(length, cudaArr, keys_d, cudaArr);    
  hipDeviceSynchronize();
  // long* h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, cudaArr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);

  //hipMemcpy(cudaArr, &first_index, sizeof(long), hipMemcpyHostToDevice);
  //hipMemcpy(cudaArr+length, &first_index, sizeof(long), hipMemcpyHostToDevice);

  //hipMemcpy(keys_d, cudaArr,  length*sizeof(long), hipMemcpyDeviceToDevice);

  thrust::inclusive_scan_by_key(thrust::cuda::par, cudaArr, cudaArr+length, cudaArr+length, cudaArr+length);

  // long* h_long_test = (long*)malloc(sizeof(long)*length*ROW2);
  // hipMemcpy(h_long_test, cudaArr, sizeof(long)*length*ROW2, hipMemcpyDeviceToHost);
  // print(h_long_test, length, ROW2);
  // free(h_long_test);

  hipMemcpy(res, cudaArr, length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);

  //double max = 0;
  //double total = 0;
  /*for(int n = nextP2*2; n>1; n=n>>1){
    //start = clock();
    propagateParentsAndCountChildrenStep<<<numBlock, BLOCKSIZE>>>(length, cudaArr, i, res);    
    hipDeviceSynchronize();
    //end = clock();
    //double time = ((double)(end-start)/CLOCKS_PER_SEC)*1000;
    //max = max > time ? max : time;
    //total += time;
    hipMemcpy(cudaArr, res,  length*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
    i+=1;
  }*/
  //printf("parents max time: %f, %d\n", max, nextP2);
  //printf("parents total time: %f\n", total);

  hipFree(cudaArr);
  hipFree(index_d);
  hipFree(keys_d);

  return res;
}

__global__
void childsNumber(int length, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(long i=index; i<length; i+=stride){
    res[i] = arr[i];
    res[length + i] = arr[length + i];
    res[length*ROW2 + i] = 0;
    if(i == length - 1) res[length*ROW2 + arr[i]] = arr[length + i];
    else if(arr[i] != arr[i + 1] && arr[i] != -1) res[length*ROW2 + arr[i]] = arr[length + i];
    if(i == 0 && length == 1) res[length*ROW2] = 0;
  }
}

__global__
void addOne(int length, long* arr)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  if(index==0) arr[length*ROW3] = 0;
  for(long i=index; i<length; i+=stride){
    arr[length*ROW3 + i] = arr[length*ROW3 + i] + 1;
  }
  if(index==0) arr[length*ROW3] = 0;
}

long * allocate(int length, int numBlock, long* arr)
{
  long * cudaArr;
  hipMalloc(&cudaArr, length*ROW4*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*ROW3*sizeof(long), hipMemcpyDeviceToDevice);
  hipMemcpy(cudaArr+length*ROW3+1, arr+length*ROW2,  (length*ROW1-1)*sizeof(long), hipMemcpyDeviceToDevice);
  addOne<<<numBlock, BLOCKSIZE>>>(length, cudaArr);
  hipDeviceSynchronize();
  thrust::inclusive_scan(thrust::cuda::par, cudaArr+ROW3*length, cudaArr + ROW4*length, cudaArr+ROW3*length);
  return cudaArr;
}

long * scan(int length, long* arr)
{
  long * cudaArr;
  long * res;
  hipMalloc(&cudaArr, length*ROW4*sizeof(long));
  hipMalloc(&res, length*ROW1*sizeof(long));
  hipMemcpy(cudaArr, arr,  length*ROW4*sizeof(long), hipMemcpyHostToDevice); 
  thrust::inclusive_scan(thrust::cuda::par, cudaArr+ROW2*length, cudaArr + ROW3*length, res);
  hipFree(cudaArr);
  return res;  
}

__global__
void generateRes(int length, long* arr, long* res)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;  
  for(long i = index; i<length; i+=stride){
    res[arr[length*ROW3 + i]] = arr[length*ROW2 + i];
    if(arr[i] != -1) res[arr[length*ROW3+ arr[i]]+arr[length+ i]] = arr[length*ROW3+ i];
  }
}


double NewRuntime_Parallel_GPU(char* input_d, int length) {
  //hipProfilerStart();
  int attachedLength = length;
  int numBlock = (attachedLength + BLOCKSIZE - 1) / BLOCKSIZE;
  long* res;
  long* fakeRes;
  long* arr;
  //char* attacheArr;
  clock_t start, end, allStart, allEnd;
  char* h_char_test;
  long* h_long_test;
  allStart = clock();

  start = clock();

  //attacheArr = input;
  //memcpy(attacheArr, input, length*sizeof(char));
  //attacheArr[length] = ',';
  char* d_attacheArr;
  hipMalloc(&d_attacheArr, attachedLength*sizeof(char));
  hipMemcpy(d_attacheArr, input_d, attachedLength*sizeof(char), hipMemcpyDeviceToDevice);
  
  //hipMallocManaged(&attacheArr, attachedLength*sizeof(char));
  //hipMemcpy(attacheArr, input, length*sizeof(char), hipMemcpyHostToDevice);
  //attacheArr[length] = ',';
  char* d_sameDepthArr;
  hipMalloc(&d_sameDepthArr, attachedLength*sizeof(char));
  hipMemcpy(d_sameDepthArr, input_d, attachedLength*sizeof(char), hipMemcpyDeviceToDevice);

  changeDepth<<<numBlock, BLOCKSIZE>>>(attachedLength, d_attacheArr, d_sameDepthArr);
  hipDeviceSynchronize();
  //free(attacheArr);
  hipFree(d_attacheArr);
  end = clock();
  step1 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------First Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // // h_char_test = (char*)malloc(sizeof(char)*attachedLength);
  // // hipMemcpy(h_char_test, d_sameDepthArr, sizeof(char)*attachedLength, hipMemcpyDeviceToHost);
  // // printString(h_char_test, attachedLength, ROW1);
  // // free(h_char_test);
  // std::cout << "-------------End First Step--------------" << std::endl;

  start = clock();
  long *d_arr;
  long correctDepth;
  correctDepth = findDepthAndCount(attachedLength, numBlock, &d_arr, d_sameDepthArr);
  end = clock();
  step2 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
  // std::cout << "-------------Second Step--------------" << std::endl;
  // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
  // // h_long_test = (long*)malloc(sizeof(long)*attachedLength*ROW3);
  // // hipMemcpy(h_long_test, d_arr, sizeof(long)*attachedLength*ROW3, hipMemcpyDeviceToHost);
  // // print(h_long_test, attachedLength, ROW3);
  // // free(h_long_test);
  // std::cout << "-------------End Second Step--------------" << std::endl;
  long arrLength;
  hipMemcpy(&arrLength, d_arr+(attachedLength-1), sizeof(long), hipMemcpyDeviceToHost);
  if(correctDepth != -1){
    bool correct;
    correct = isCorrect(attachedLength, d_arr+(attachedLength)*ROW2, d_sameDepthArr);
    if(correct){      
      start = clock();
      hipMalloc(&arr, attachedLength*ROW4*sizeof(long));
      hipMalloc(&res, arrLength*ROW4*sizeof(long));
      hipMemcpy(arr, d_arr,  attachedLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      reduce<<<numBlock, BLOCKSIZE>>>(attachedLength, arrLength, d_sameDepthArr, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step3 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Third Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW2);
      // // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW2);
      // // free(h_long_test);
      // std::cout << "-------------End Third Step--------------" << std::endl;
      hipFree(d_sameDepthArr);
      hipFree(d_arr);
      int numBlock = (arrLength + BLOCKSIZE - 1) / BLOCKSIZE;

      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      fakeRes = sortByDepth(arrLength, numBlock, arr);
      end = clock();
      step4 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Fourth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long *)malloc(sizeof(long)*arrLength*ROW2);
      // // hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW2);
      // // free(h_long_test);
      // std::cout << "-------------End Fourth Step--------------" << std::endl;

      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      hipFree(fakeRes);
      hipMemset(res, -1, arrLength*ROW1*sizeof(long));
      findParents<<<numBlock, BLOCKSIZE>>>( arrLength, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step5 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Fifth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW1);
      // // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW1, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW1);
      // // free(h_long_test);
      // std::cout << "-------------End Fifth Step--------------" << std::endl;
      
      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW1*sizeof(long), hipMemcpyDeviceToDevice);
      fakeRes = propagateParentsAndCountChildren(arrLength, numBlock, arr);
      end = clock();
      step6 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Sixth Step--------------" << std::endl;      
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // //  h_long_test = (long*)malloc(sizeof(long)*arrLength*ROW2);
      // //  hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW2, hipMemcpyDeviceToHost);
      // //  print(h_long_test, arrLength, ROW2);
      // //  free(h_long_test);
      // std::cout << "-------------End Sixth Step--------------" << std::endl;

      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW2*sizeof(long), hipMemcpyDeviceToDevice);
      hipFree(fakeRes);
      hipMemset(res, -1, arrLength*ROW3*sizeof(long));
      childsNumber<<<numBlock, BLOCKSIZE>>>(arrLength, arr, res);
      hipDeviceSynchronize();
      end = clock();
      step7 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Seventh Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW3);
      // // hipMemcpy(h_long_test, res, sizeof(long)*arrLength*ROW3, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW3);
      // // free(h_long_test);
      // std::cout << "-------------End Seventh Step--------------" << std::endl;
      
      start = clock();
      hipMemcpy(arr, res,  arrLength*ROW3*sizeof(long), hipMemcpyDeviceToDevice);
      fakeRes = allocate(arrLength, numBlock, arr);
      end = clock();
      step8 += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Eighth Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW4);
      // // hipMemcpy(h_long_test, fakeRes, sizeof(long)*arrLength*ROW4, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW4);
      // // free(h_long_test);
      // std::cout << "-------------End Eighth Step--------------" << std::endl;

      long* sumRes;
      //hipMalloc(&sumRes, arrLength*ROW1*sizeof(long));
      start = clock();
      sumRes = scan(arrLength, fakeRes);
      end = clock();
      scanStep += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Scan Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // // h_long_test = (long*) malloc(sizeof(long)*arrLength*ROW1);
      // // hipMemcpy(h_long_test, sumRes, sizeof(long)*arrLength*ROW1, hipMemcpyDeviceToHost);
      // // print(h_long_test, arrLength, ROW1);
      // // free(h_long_test);
      // std::cout << "-------------End Scan Step--------------" << std::endl;
      long resLength;
      hipMemcpy(&resLength, sumRes + arrLength - 1, sizeof(long), hipMemcpyDeviceToHost);
      start = clock();
      hipMemcpy(arr, fakeRes,  arrLength*ROW4*sizeof(long), hipMemcpyHostToHost);
      hipFree(sumRes);
      hipFree(fakeRes);
      hipFree(res);
      hipMalloc(&res, (arrLength+resLength)*sizeof(long));
      hipMemset(res, 0, (arrLength+resLength)*sizeof(long));
      generateRes<<<numBlock, BLOCKSIZE>>>(arrLength,  arr, res);
      hipDeviceSynchronize();
      end = clock();
      lastStep += ((double)(end-start)/CLOCKS_PER_SEC)*1000;
      // std::cout << "-------------Last Step--------------" << std::endl;
      // std::cout << "Time elapsed: " << std::setprecision (17) << ((double)(end-start)/CLOCKS_PER_SEC)*1000 << std::endl;
      // //h_long_test = (long*) malloc(sizeof(long)*(arrLength+resLength)*ROW1);
      // //hipMemcpy(h_long_test, res, sizeof(long)*(arrLength+resLength)*ROW1, hipMemcpyDeviceToHost);
      // //print(h_long_test, (arrLength+resLength), ROW1);
      // //free(h_long_test);
      // std::cout << "-------------End Last Step--------------" << std::endl;

      hipFree(arr);
      hipFree(res);
      allEnd = clock();
      //hipProfilerStop();    
      //*******************************//
      // size_t l_free = 0;
      // size_t l_Total = 0;
      // hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
      // size_t allocated = (l_Total - l_free);
      // std::cout << "Total: " << l_Total << " Free: " << l_free << " Allocated: " << allocated << std::endl;
      //*******************************//
      //program += ((double)(allEnd-allStart)/CLOCKS_PER_SEC)*1000;
      //printf("program: %f\n", program);

    } 
    else{
      printf("Input wrong\n");
      return 0;  
    }
  }
  else {
    printf("Input invalid\n");
    return 0;
  }
  return (double)(allEnd-allStart);
}
